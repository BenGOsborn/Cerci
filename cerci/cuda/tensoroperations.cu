#include "hip/hip_runtime.h"
#include "tensoroperations.cuh"

const int THREAD_SIZE_XY = 1 << 10;
const int THREAD_SIZE_Z = 1 << 6;

__global__
void addElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] + ptr2[idx];
}

std::unique_ptr<float[]> CUDAaddElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    addElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void subtractElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] - ptr2[idx];
}

std::unique_ptr<float[]> CUDAsubtractElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    subtractElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void multiplyElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] * ptr2[idx];
}

std::unique_ptr<float[]> CUDAmultiplyElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    multiplyElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void divideElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] / ptr2[idx];
}

std::unique_ptr<float[]> CUDAdivideElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    divideElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void powerElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = std::pow(ptr1[idx], ptr2[idx]);
}

std::unique_ptr<float[]> CUDApowerElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    powerElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void transposeD(int rows, int cols, int depths, float* ptr1, float* ptr2) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int depth = blockIdx.z * blockDim.z + threadIdx.z;
    // Of course this is going to need a Z coordinate for the infinite dimensions it can take
    if ((depth < depths) && (row < rows) && (col < cols)) ptr2[depth * rows * cols + row * cols + col] = ptr1[depth * rows * cols + col * rows + row];
}

// I need to reformat all of the other functions to fit this
std::unique_ptr<float[]> CUDAtranspose(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<int[]>& in_ptr1_dims, int in_ptr1_dims_size, int ptr1_size) {
    int bytes = ptr1_size * sizeof(float);

    int cols = in_ptr1_dims[0];
    int rows = in_ptr1_dims[1];
    // Is there a faster way to do this
    int depths = 1;
    for (int i = 2; i < in_ptr1_dims_size; i++) {
        depths *= in_ptr1_dims[i];
    }

    // The problem is thats the max amount of threads that can be launched Im assuming but Im multiplying it by 3 for each dimension
    // Leading to there being more dimensions
    float* gpu_ptr1;
    float* gpu_ptr2;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);

    // And of course it just failed because I just overloaded its size of threads allowed
    int grid_cols = (cols + std::sqrt(THREAD_SIZE_XY / THREAD_SIZE_Z) - 1) / std::sqrt(THREAD_SIZE_XY / THREAD_SIZE_Z);
    int grid_rows = (rows + std::sqrt(THREAD_SIZE_XY / THREAD_SIZE_Z) - 1) / std::sqrt(THREAD_SIZE_XY / THREAD_SIZE_Z);
    int grid_depths = (depths + THREAD_SIZE_Z - 1) / THREAD_SIZE_Z;

    dim3 gridSize(grid_cols, grid_cols, grid_depths);
    dim3 threadSize(std::sqrt(THREAD_SIZE_XY / THREAD_SIZE_Z), std::sqrt(THREAD_SIZE_XY / THREAD_SIZE_Z), THREAD_SIZE_Z);

    transposeD <<< gridSize, threadSize >>> (rows, cols, depths, gpu_ptr1, gpu_ptr2);

    std::unique_ptr<float[]> out_ptr(new float[ptr1_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr2, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
   
    return out_ptr;
} 