
#include <hip/hip_runtime.h>
int main() {
    int N = 1<<2;
    size_t bytes = N*sizeof(int);

    float *x; 
    float *y;

    hipMallocManaged(&x, bytes);
    hipMallocManaged(&y, bytes);

    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
    }

    hipFree(x);
    hipFree(y);

    return 0;
}