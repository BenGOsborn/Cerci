#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <stdexcept>

class Matrix {
	private:
		float* matrix;
		unsigned int* size;
		unsigned int* shape;

	public:
		// This shape and matrix being parsed seperaretly will probably have to be changed at some point in time or we can just keep it like this?
		Matrix(float *inMatrix, unsigned int *inShape) {

			shape = (unsigned int*)malloc(3 * sizeof(unsigned int));
			memcpy(shape, inShape, 3 * sizeof(unsigned int));

			size = new unsigned int(shape[0]*shape[1]);
			
			matrix = (float*)malloc(*size * sizeof(float));
			memcpy(matrix, inMatrix, *size * sizeof(float));
		}

		void print() {
			for (int i = 0; i < shape[0]; i++) {
				for (int j = 0; j < shape[1]; j++) {
					std::cout << matrix[i*shape[1]+j] << " ";
				}
				std::cout << "\n";
			}
		}

		Matrix* reshape(unsigned int rows, unsigned int cols) {
			if (rows * cols != *size) throw std::invalid_argument("New matrix size does not match original matrix size!");

			unsigned int* new_shape;
			new_shape = (unsigned int*)malloc(2 * sizeof(unsigned int));
			new_shape[0] = rows;
			new_shape[1] = cols;

			Matrix* ret_matrix = new Matrix(matrix, new_shape);

			free(new_shape);

			return ret_matrix;
		}

		// This could be done in parallel on the GPU
		Matrix* transpose() {
			unsigned int* new_shape;
			new_shape = (unsigned int*)malloc(2 * sizeof(unsigned int));	
			new_shape[0] = shape[1];
			new_shape[1] = shape[0];

			float* new_matrix;
			new_matrix = (float*)malloc(*size * sizeof(float));
			for (int i = 0; i < shape[0]; i++) {
				for (int j = 0; j < shape[1]; j++) {
					new_matrix[j * new_shape[1] + i] = matrix[i * shape[1] + j];
				}
			}

			Matrix* ret_matrix = new Matrix(new_matrix, new_shape);
			
			free(new_shape);
			free(new_matrix);

			return ret_matrix;
		}

		Matrix* clone() {
			Matrix* ret_matrix = new Matrix(matrix, shape);
			return ret_matrix;
		}

		// Functions like this can probably be done in parallel on the GPU
		Matrix* applyFunc(float(*func)(float)) {
			float* new_matrix;
			new_matrix = (float*)malloc(*size * sizeof(float));

			for (int i = 0; i < *size; i++) {
				new_matrix[i] = func(new_matrix[i]);
			}

			Matrix* ret_matrix = new Matrix(new_matrix, shape);

			free(new_matrix);

			return ret_matrix;
		}

		float* returnMatrix() {
			float* ret_matrix;
			ret_matrix = (float*)malloc(*size * sizeof(float));
			memcpy(ret_matrix, matrix, *size * sizeof(float));

			return ret_matrix;
		}

		unsigned int* returnShape() {
			return shape;
		}

		unsigned int returnSize() {
			return *size;
		}

		~Matrix() {
			free(matrix);
			free(size);
			free(shape);
		}
};

__global__
void addMatricesD(int size, float* vector1, float *vector2, float *retVector) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) retVector[idx] = vector1[idx] + vector2[idx];
}

// For subtraction just multiply the array by 1 the add into a negative which can be done through the apply func
Matrix* addMatrices(Matrix *matrix1, Matrix *matrix2) {
	if (matrix1->returnShape() != matrix2->returnShape()) throw std::invalid_argument("Matrices are not of the same shape!");

	unsigned int size = matrix1->returnSize();
	int bytes = size * sizeof(float);

	float* mat1 = matrix1->returnMatrix();
	float* mat2 = matrix2->returnMatrix();
	float* mat3;
	mat3 = (float*)malloc(bytes);

	float* mat1d;
	float* mat2d;
	float* mat3d;
	hipMalloc(&mat1d, bytes);
	hipMalloc(&mat2d, bytes);
	hipMalloc(&mat3d, bytes);
	hipMemcpy(mat1d, mat1, bytes, hipMemcpyHostToDevice);
	hipMemcpy(mat2d, mat2, bytes, hipMemcpyHostToDevice);

	int NUM_THREADS = 1 << 10;
	int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;

	addVectors <<< NUM_BLOCKS, NUM_THREADS >>> (size, mat1d, mat2d, mat3d);

	hipMemcpy(mat3, mat3d, bytes, hipMemcpyDeviceToHost);

	unsigned int* shape = matrix1->returnShape();
	Matrix* ret_matrix = new Matrix(mat3, shape);

	// What pointers are the ones that I need to free up here?
	hipFree(mat1d);
	hipFree(mat2d);
	hipFree(mat3d);
	free(shape);
	free(mat1);
	free(mat2);
	free(mat3);

	return ret_matrix;
}

__global__
void multiplyMatricesD() {
	
}

Matrix* multiplyMatrices(Matrix *matrix1, Matrix *matrix2) {
	if (matrix1->returnShape()[1] != matrix2->returnShape()[0]) throw std::invalid_argument("Matrix dimensions are not aligned for multiplication!");

	
}

int main() {
	unsigned int* shape;
	shape = (unsigned int*)malloc(2 * sizeof(unsigned int));
	shape[0] = 5;
	shape[1] = 2;

	float* vals;
	vals = (float*)malloc(10 * sizeof(float));
	for (int i = 0; i < 10; i++) {
		vals[i] = 1.0f;
	}

	Matrix* matrix = new Matrix(vals, shape);
	Matrix* transposed = matrix->transpose();
	transposed->print();

	delete matrix;
	delete transposed;
	free(shape);
	free(vals);
}