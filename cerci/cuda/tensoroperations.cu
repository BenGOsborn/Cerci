#include "hip/hip_runtime.h"
#include "tensoroperations.cuh"

__global__
void addD(int size, int& ptr1, int& ptr2) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
}

std::unique_ptr<float[]> CUDAaddElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<int>& in_ptr1_dims, int dim_size1, std::unique_ptr<float[]>& in_ptr2, std::unique_ptr<int>& in_ptr2_dims, int dim_size2) {
    // We could just assume that they are of the same length...?
    // Its cheaper to do the error checking here than to do it on init of each tensor

    if (dim_size1 != dim_size2) throw std::invalid_argument("Pointers are of different sizes!");

    // I could do a try statement here which checks outside of the lengths of the dimsize to see if it can go any further, then if it can returns failure
    bool valid = false;
    try {
        in_ptr1_dims[dim_size1];
        valid = false;
        break;
        in_ptr2_dims[dim_size1];
        valid = false;
    } catch (const std::exception& e) { 
        // This is a bit broken actually
        throw std::invalid_argument("Sizes do not match the pointer size!");
    }

    int size;
    for (int i = 0; i < dim_size1; i++) {
       if (in_ptr1_dims[i] != in_ptr2_dims[i]) throw std::invalid_argument("Dimensions are of different sizes!"); 
       size *= in_ptr1_dims[i];
    }

    int bytes = size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (size + THREAD_SIZE - 1) / THREAD_SIZE;
    addD <<< dimGradX, THREAD_SIZE >>> (size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unqiue_ptr<float[]> out_ptr3(new float[int_ptr1_size]);
    hipMemcpy(out_ptr3.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr3;
}