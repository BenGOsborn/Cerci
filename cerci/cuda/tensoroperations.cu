#include "hip/hip_runtime.h"
#include "tensoroperations.cuh"

__global__
void addD(int size, int& ptr1, int& ptr2) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
}

std::unique_ptr<float[]> CUDAaddElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int size) {
    // We could just assume that they are of the same length...?
    // Its cheaper to do the error checking here than to do it on init of each tensor
    // This assumes that the values we give it are correct as there is no error checking
    // Remove all checking

    int bytes = size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (size + THREAD_SIZE - 1) / THREAD_SIZE;
    addD <<< dimGradX, THREAD_SIZE >>> (size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unqiue_ptr<float[]> out_ptr3(new float[int_ptr1_size]);
    hipMemcpy(out_ptr3.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr3;
}