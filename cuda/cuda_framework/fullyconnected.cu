#include "fullyconnected.cuh"

FullyConnected::FullyConnected(std::unique_ptr<Matrix>& weight_set, std::unique_ptr<Matrix>& bias_set, float lr = 0.1) {
	FullyConnected::weights = weight_set->clone();
	FullyConnected::bias = bias_set->clone();
	FullyConnected::learning_rate = lr;
}

std::unique_ptr<Matrix> FullyConnected::predict(std::unique_ptr<Matrix>& inputs) {
	std::unique_ptr<Matrix> multiplied = multiply(FullyConnected::weights, inputs);
	std::unique_ptr<Matrix> out = add(multiplied, FullyConnected::bias);

	return out;
}

std::unique_ptr<Matrix> FullyConnected::train(std::unique_ptr<Matrix>& inputs, std::unique_ptr<Matrix>& errors) {
	std::unique_ptr<Matrix> inputs_transposed = inputs->transpose();
	std::unique_ptr<Matrix> weight_adjustments = multiply(errors, inputs_transposed);

	std::unique_ptr<Matrix> weights_lr = multiplyScalar(weight_adjustments, FullyConnected::learning_rate);
	std::unique_ptr<Matrix> bias_lr = multiplyScalar(errors, FullyConnected::learning_rate);

	FullyConnected::weights = subtract(FullyConnected::weights, weights_lr);
	FullyConnected::bias = subtract(FullyConnected::bias, bias_lr);

	std::unique_ptr<Matrix> weights_transposed = FullyConnected::weights->transpose();
	std::unique_ptr<Matrix> back_errors = multiply(errors, weights_transposed);

	return back_errors;
}

// This function should only ever be used for deepQlearning where two models are required
void FullyConnected::setNetwork(std::unique_ptr<Matrix>& new_weights, std::unique_ptr<Matrix>& new_bias) {
	FullyConnected::weights = new_weights->clone();
	FullyConnected::bias = new_bias->clone();
}
