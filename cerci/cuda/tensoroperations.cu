#include "hip/hip_runtime.h"
#include "tensoroperations.cuh"

const int THREAD_SIZE = 1 << 10;
const int BLOCK_SIZE = 1 << 5;

__global__
void addElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] + ptr2[idx];
}

std::unique_ptr<float[]> CUDAaddElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE - 1) / THREAD_SIZE;
    addElementwiseD <<< dimGridX, THREAD_SIZE >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr3(new float[ptr_size]);
    hipMemcpy(out_ptr3.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr3;
}

__global__
void subtractElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] - ptr2[idx];
}

std::unique_ptr<float[]> CUDAsubtractElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE - 1) / THREAD_SIZE;
    subtractElementwiseD <<< dimGridX, THREAD_SIZE >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr3(new float[ptr_size]);
    hipMemcpy(out_ptr3.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr3;
}

__global__
void multiplyElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] * ptr2[idx];
}

std::unique_ptr<float[]> CUDAmultiplyElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE - 1) / THREAD_SIZE;
    multiplyElementwiseD <<< dimGridX, THREAD_SIZE >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr3(new float[ptr_size]);
    hipMemcpy(out_ptr3.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr3;
}

__global__
void divideElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] / ptr2[idx];
}

std::unique_ptr<float[]> CUDAdivideElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE - 1) / THREAD_SIZE;
    divideElementwiseD <<< dimGridX, THREAD_SIZE >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr3(new float[ptr_size]);
    hipMemcpy(out_ptr3.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr3;
}

__global__
void powerElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = std::pow(ptr1[idx], ptr2[idx]);
}

std::unique_ptr<float[]> CUDApowerElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    // We could just assume that they are of the same length...?
    // Its cheaper to do the error checking here than to do it on init of each tensor
    // This assumes that the values we give it are correct as there is no error checking
    // Remove all checking
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE - 1) / THREAD_SIZE;
    powerElementwiseD <<< dimGridX, THREAD_SIZE >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr3(new float[ptr_size]);
    hipMemcpy(out_ptr3.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr3;
}