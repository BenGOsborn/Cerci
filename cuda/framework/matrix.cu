
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"\nGPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void vectorAdd(int n, int *x, int *y) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < n) y[i] = x[i] + y[i];
}

int main(void) {
    int N = 1<<4;
    int bytes = N*sizeof(int);

    // Allocate memory on the host device
    int *x, *y;
    x = (int*)malloc(bytes);
    y = (int*)malloc(bytes);

    // Allocate mrmoy on the GPU
    int *d_x, *d_y;
    hipMalloc(&d_x, bytes); 
    hipMalloc(&d_y, bytes);

    // Initialize values on the host device
    for (int i = 0; i < N; i++) {
        x[i] = rand() % 100;
        y[i] = rand() % 100;
    }

    std::cout << "Initial x: ";
    for (int i = 0; i < N; i++) {
        std::cout << x[i] << " ";
    }
    std::cout << "\nInitial y: ";
    for (int i = 0; i < N; i++) {
        std::cout << y[i] << " ";
    }
    std::cout << std::endl;

    // Copy the memory from the host device to the GPU
    hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements on the GPU
    vectorAdd<<<(N+255)/256, 256>>>(N, d_x, d_y);

    gpuErrchk( hipPeekAtLastError() );

    hipMemcpy(y, d_y, bytes, hipMemcpyDeviceToHost);

    std::cout << "Final: ";
    for (int i = 0; i < N; i++) {
        std::cout << y[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
}