#include "hip/hip_runtime.h"
#include "tensoroperations.cuh"

const int THREAD_SIZE_XY = 1 << 10;
const int THREAD_SIZE_Z = 1 << 6;

__global__
void addElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] + ptr2[idx];
}

std::unique_ptr<float[]> CUDAaddElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    addElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void subtractElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] - ptr2[idx];
}

std::unique_ptr<float[]> CUDAsubtractElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    subtractElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void multiplyElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] * ptr2[idx];
}

std::unique_ptr<float[]> CUDAmultiplyElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    multiplyElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void divideElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = ptr1[idx] / ptr2[idx];
}

std::unique_ptr<float[]> CUDAdivideElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    divideElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
void powerElementwiseD(int size, float* ptr1, float* ptr2, float* ptr3) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) ptr3[idx] = std::pow(ptr1[idx], ptr2[idx]);
}

std::unique_ptr<float[]> CUDApowerElementwise(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<float[]>& in_ptr2, int ptr_size) {
    int bytes = ptr_size * sizeof(float);

    float* gpu_ptr1;
    float* gpu_ptr2;
    float* gpu_ptr3;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);
    hipMalloc(&gpu_ptr3, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(gpu_ptr2, in_ptr2.get(), bytes, hipMemcpyHostToDevice);

    int dimGridX = (ptr_size + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    powerElementwiseD <<< dimGridX, THREAD_SIZE_XY >>> (ptr_size, gpu_ptr1, gpu_ptr2, gpu_ptr3);

    std::unique_ptr<float[]> out_ptr(new float[ptr_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr3, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
    hipFree(gpu_ptr3);
   
    return out_ptr;
}

__global__
// I might need a seperate pointer for this because it would of overwritten the value
void transposeD(int rows, int cols, int depths, float* ptr1, float* ptr2) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int depth = blockIdx.z * blockDim.z + threadIdx.z;
    // Of course this is going to need a Z coordinate for the infinite dimensions it can take
    if ((depth < depths) && (row < rows) && (col < cols)) ptr2[depth * rows * cols + row * cols + col] = ptr1[depth * rows * cols + col * rows + row];
}

// I need to reformat all of the other functions to fit this
std::unique_ptr<float[]> CUDAtranspose(std::unique_ptr<float[]>& in_ptr1, std::unique_ptr<int[]>& in_ptr1_dims, int in_ptr1_dims_size, int ptr1_size) {
    int bytes = ptr1_size * sizeof(float);

    int cols = in_ptr1_dims[0];
    int rows = in_ptr1_dims[1];
    // Is there a faster way to do this
    int depths = 1;
    for (int i = 2; i < in_ptr1_dims_size; i++) {
        depths *= in_ptr1_dims[i];
    }

    float* gpu_ptr1;
    float* gpu_ptr2;
    hipMalloc(&gpu_ptr1, bytes);
    hipMalloc(&gpu_ptr2, bytes);

    hipMemcpy(gpu_ptr1, in_ptr1.get(), bytes, hipMemcpyHostToDevice);




    // We divide it because this is how many blocks we need to have and then there is BLOCK_SIZE num of threads per block
    // It is failing because I am trying to launch too many threads at once
    int grid_cols = (cols + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY; 
    int grid_rows = (rows + THREAD_SIZE_XY - 1) / THREAD_SIZE_XY;
    int grid_depths = (depths + THREAD_SIZE_Z - 1) / THREAD_SIZE_Z;

    std::cout << THREAD_SIZE_XY << " " << THREAD_SIZE_Z << std::endl; // This is the size of the thread sizes
    std::cout << grid_cols << " " << grid_rows << " " << grid_depths << std::endl; // This is the size of the blocks
    std::cout << grid_cols*THREAD_SIZE_XY << " " << grid_rows*THREAD_SIZE_XY << " " << grid_depths*THREAD_SIZE_Z << std::endl; // This is the size of the threads launched

    dim3 dimGrid(grid_cols, grid_rows, grid_depths);
    dim3 dimThreads(THREAD_SIZE_XY, THREAD_SIZE_XY, THREAD_SIZE_Z);
    
    transposeD <<< dimGrid, dimThreads >>> (cols, rows, depths, gpu_ptr1, gpu_ptr2);
    cudaErr( hipPeekAtLastError() );


    

    // Change all of these from out_ptr3 to out_ptr as required
    std::unique_ptr<float[]> out_ptr(new float[ptr1_size]);
    hipMemcpy(out_ptr.get(), gpu_ptr2, bytes, hipMemcpyDeviceToHost);

    hipFree(gpu_ptr1);
    hipFree(gpu_ptr2);
   
    return out_ptr;
} 