#include "dropout.cuh"

Dropout::Dropout(float rate) {
	if ((rate < 0) || (rate >= 1)) throw std::invalid_argument("Rate must be >= 0 and < 1!");
	Dropout::dropout_rate = rate;
}

std::unique_ptr<Matrix> Dropout::applyDropout(std::unique_ptr<Matrix>& predictions) {
	// So first we want to create a mask with the same shape as in the input 
	int size = predictions->returnSize();
	std::unique_ptr<int[]> pred_shape = predictions->returnShape();
	std::unique_ptr<float[]> vals = std::make_unique<float[]>(size);
	for (int i = 0; i < size; i++) {
		if (1.0 * (rand() % 1000) / 1000 < Dropout::dropout_rate) {
			vals[i] = 1.0 / (1 - Dropout::dropout_rate);
		}
		else {
			vals[i] = 0.0f;
		}
	}
	std::unique_ptr<Matrix> dropout_mask = std::make_unique<Matrix>(vals, pred_shape);
	// Is it needed to have all of these clones here?
	Dropout::mask = dropout_mask->clone();
	std::unique_ptr<Matrix> mask_applied = multiplyElementwise(predictions, dropout_mask);

	return mask_applied;
}

std::unique_ptr<Matrix> Dropout::backErrors(std::unique_ptr<Matrix>& errors) {
	std::unique_ptr<Matrix> applied = multiplyElementwise(errors, Dropout::mask);
	
	return applied;
}
