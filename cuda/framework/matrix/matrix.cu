#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <stdexcept>

int NUM_THREADS = 1 << 10;

template <typename Lambda>
__global__ 
void applyD(int size, float* inVector, Lambda function) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) inVector[idx] = function(inVector[idx]);
}

__global__
void transposeD(int rows, int cols, float* inVector) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	if ((idy < rows) && (idx < cols)) inVector[idx * cols + idy] = inVector[idy * rows + idx];
}

class Matrix {
private:
	std::unique_ptr<float[]> matrix;
	std::unique_ptr<int> size;
	std::unique_ptr<int[]> shape;

	// If I remove the macro then I should redefine the 'NUM_THREADS'
	std::unique_ptr<int> NUM_BLOCKS;

public:
	Matrix(std::unique_ptr<float[]>& inMatrix, std::unique_ptr<int[]>& inShape) {
		shape = std::make_unique<int[]>(2);
		memcpy(shape.get(), inShape.get(), 2 * sizeof(int));

		size = std::make_unique<int>(shape[0] * shape[1]);

		matrix = std::make_unique<float[]>(*size);
		memcpy(matrix.get(), inMatrix.get(), *size * sizeof(float));

		// This constant 'NUM_THREADS' can be adjusted
		NUM_BLOCKS = std::make_unique<int>((*size + NUM_THREADS - 1) / NUM_THREADS);
	}

	void print() {
		for (int i = 0; i < shape[0]; i++) {
			for (int j = 0; j < shape[1]; j++) {
				std::cout << matrix[i * shape[1] + j] << " ";
			}
			std::cout << "\n";
		}
	}

	std::unique_ptr<Matrix> reshape(int rows, int cols) {
		if (rows * cols != *size) throw std::invalid_argument("New matrix size does not match original matrix size!");

		std::unique_ptr<int[]> new_shape = std::make_unique<int[]>(2);
		new_shape[0] = rows;
		new_shape[1] = cols;

		std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(matrix, new_shape);

		return ret_matrix;
	}

	// This could be done in parallel on the GPU
	std::unique_ptr<Matrix> transpose() {
		std::unique_ptr<int[]> new_shape = std::make_unique<int[]>(2);
		new_shape[0] = shape[1];
		new_shape[1] = shape[0];

		int bytes = *size * sizeof(float);

		float* dCopy;
		hipMalloc(&dCopy, bytes);
		hipMemcpy(dCopy, matrix.get(), bytes, hipMemcpyHostToDevice);

		transposeD <<< *NUM_BLOCKS, NUM_THREADS >>> (shape[0], shape[1], dCopy);

		std::unique_ptr<float[]> new_matrix = std::make_unique<float[]>(*size);
		hipMemcpy(new_matrix.get(), dCopy, bytes, hipMemcpyDeviceToHost);

		std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(new_matrix, new_shape);

		hipFree(dCopy);

		return ret_matrix;
	}

	std::unique_ptr<Matrix> clone() {
		std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(matrix, shape);
		return ret_matrix;
	}

	// I cant run this on my GPU yet for some reason so it has to be done on the CPU I will get it working though
	template <typename Lambda>
	std::unique_ptr<Matrix> apply(Lambda function) {
		int bytes = *size * sizeof(float);

		float* dCopy;
		hipMalloc(&dCopy, bytes);
		hipMemcpy(dCopy, matrix.get(), bytes, hipMemcpyHostToDevice);

		applyD <<< *NUM_BLOCKS, NUM_THREADS >>> (*size, dCopy, function);

		std::unique_ptr<float[]> new_matrix = std::make_unique<float[]>(*size);
		hipMemcpy(new_matrix.get(), dCopy, bytes, hipMemcpyDeviceToHost);

		std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(new_matrix, shape);

		hipFree(dCopy);

		return ret_matrix;
	}

	std::unique_ptr<float[]> returnMatrix() {
		std::unique_ptr<float[]> ret_matrix = std::make_unique<float[]>(*size);
		memcpy(ret_matrix.get(), matrix.get(), *size * sizeof(float));

		return ret_matrix;
	}

	std::unique_ptr<int[]> returnShape() {
		std::unique_ptr<int[]> ret_shape = std::make_unique<int[]>(2);
		memcpy(ret_shape.get(), shape.get(), 2 * sizeof(unsigned int));
		return ret_shape;
	}

	int returnSize() {
		return *size;
	}
};

__global__
void addD(int size, float* vector1, float *vector2, float *retVector) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) retVector[idx] = vector1[idx] + vector2[idx];
}

// For subtraction just multiply the array by 1 the add into a negative which can be done through the apply func
std::unique_ptr<Matrix> add(std::unique_ptr<Matrix> &matrix1, std::unique_ptr<Matrix> &matrix2) {
	std::unique_ptr<int[]> mat1shape = matrix1->returnShape();
	std::unique_ptr<int[]> mat2shape = matrix2->returnShape();
	if ((mat1shape[0] != mat2shape[0]) || (mat1shape[1] != mat2shape[1])) throw std::invalid_argument("Matrices are not of the same shape!");

	int size = matrix1->returnSize();
	int bytes = size * sizeof(float);

	std::unique_ptr<float[]> mat1 = matrix1->returnMatrix();
	std::unique_ptr<float[]> mat2 = matrix2->returnMatrix();

	float* mat1d;
	float* mat2d;
	float* mat3d;
	hipMalloc(&mat1d, bytes);
	hipMalloc(&mat2d, bytes);
	hipMalloc(&mat3d, bytes);
	hipMemcpy(mat1d, mat1.get(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(mat2d, mat2.get(), bytes, hipMemcpyHostToDevice);

	int NUM_BLOCKS = (size + NUM_THREADS - 1) / NUM_THREADS;

	addD <<< NUM_BLOCKS, NUM_THREADS >>> (size, mat1d, mat2d, mat3d);

	std::unique_ptr<float[]> mat3 = std::make_unique<float[]>(bytes);
	hipMemcpy(mat3.get(), mat3d, bytes, hipMemcpyDeviceToHost);

	std::unique_ptr<int[]> shape = matrix1->returnShape();
	std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(mat3, shape);

	hipFree(mat1d);
	hipFree(mat2d);
	hipFree(mat3d);

	return ret_matrix;
}

__global__
// What are the specifications required for matrix multiplication...?
void multiplyD(int m, int n, int k, float* vector1, float* vector2, float* retVector) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if ((col < k) && (row < m)) {
		float sum = 0;
		for (int i = 0; i < n; i++) {
			sum += vector1[row * n + i] * vector2[i * k + col];
		}
		retVector[row * k + col] = sum;
	}
}

std::unique_ptr<Matrix> multiply(std::unique_ptr<Matrix> &matrix1, std::unique_ptr<Matrix> &matrix2) {
	std::unique_ptr<int[]> mat1shape = matrix1->returnShape();
	std::unique_ptr<int[]> mat2shape = matrix2->returnShape();
	if (mat1shape[1] != mat2shape[0]) throw std::invalid_argument("Matrix1's cols must equal Matrix2's rows!");

	std::unique_ptr<int[]> new_shape = std::make_unique<int[]>(2);
	new_shape[0] = mat1shape[0];
	new_shape[1] = mat2shape[1];
	int same = mat1shape[1];

	int new_size = new_shape[0] * new_shape[1];

	int mat1bytes = matrix1->returnSize() * sizeof(float);
	int mat2bytes = matrix2->returnSize() * sizeof(float);
	int mat3bytes = new_size * sizeof(float);

	float* mat1d;
	float* mat2d;
	float* mat3d;
	hipMalloc(&mat1d, mat1bytes);
	hipMalloc(&mat2d, mat2bytes);
	hipMalloc(&mat3d, mat3bytes);

	std::unique_ptr<float[]> mat1 = matrix1->returnMatrix();
	std::unique_ptr<float[]> mat2 = matrix2->returnMatrix();
	hipMemcpy(mat1d, mat1.get(), mat1bytes, hipMemcpyHostToDevice);
	hipMemcpy(mat2d, mat2.get(), mat2bytes, hipMemcpyHostToDevice);

	int NUM_BLOCKS = (new_size + NUM_THREADS - 1) / NUM_THREADS;
	multiplyD <<< NUM_BLOCKS, NUM_THREADS >>> (new_shape[0], same, new_shape[1], mat1d, mat2d, mat3d);

	std::unique_ptr<float[]> mat3 = std::make_unique<float[]>(new_size);
	hipMemcpy(mat3.get(), mat3d, mat3bytes, hipMemcpyDeviceToHost);

	std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(mat3, new_shape);

	hipFree(mat1d);
	hipFree(mat2d);
	hipFree(mat3d);

	return ret_matrix;
}

// Make some functions which can initialize matrices for us now

int main() {
	std::unique_ptr<int[]> shape1 = std::make_unique<int[]>(2);
	shape1[0] = 2;
	shape1[1] = 2;
	std::unique_ptr<float[]> vals1 = std::make_unique<float[]>(10);
	for (int i = 0; i < 10; i++) {
		vals1[i] = 2.0f;
	}
	std::unique_ptr<Matrix> matrix1 = std::make_unique<Matrix>(vals1, shape1);

	std::unique_ptr<int[]> shape2 = std::make_unique<int[]>(2);
	shape2[0] = 2;
	shape2[1] = 2;
	std::unique_ptr<float[]> vals2 = std::make_unique<float[]>(10);
	for (int i = 0; i < 10; i++) {
		vals2[i] = 3.0f;
	}
	std::unique_ptr<Matrix> matrix2 = std::make_unique<Matrix>(vals2, shape2);

	std::unique_ptr<Matrix> multiplied = multiply(matrix1, matrix2);
	multiplied->print();
}
