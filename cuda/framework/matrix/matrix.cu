#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <stdexcept>

int NUM_THREADS = 1 << 10;
int NUM_THREADS_BLOCK = 1 << 5;

template <typename Lambda>
__global__ 
void applyD(int size, float* inVector, Lambda function) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < size) inVector[index] = function(inVector[index]);
}

// Do I have to add multiple thread blocks here too?
__global__
void transposeD(int rows, int cols, float* inVector) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	if ((row < rows) && (col < cols)) inVector[col * cols + row] = inVector[row * rows + col];
}

class Matrix {
private:
	std::unique_ptr<float[]> matrix;
	std::unique_ptr<int> size;
	std::unique_ptr<int[]> shape;

public:
	Matrix(std::unique_ptr<float[]>& inMatrix, std::unique_ptr<int[]>& inShape) {
		shape = std::make_unique<int[]>(2);
		memcpy(shape.get(), inShape.get(), 2 * sizeof(int));

		size = std::make_unique<int>(shape[0] * shape[1]);

		matrix = std::make_unique<float[]>(*size);
		memcpy(matrix.get(), inMatrix.get(), *size * sizeof(float));
	}

	void print() {
		for (int i = 0; i < shape[0]; i++) {
			for (int j = 0; j < shape[1]; j++) {
				std::cout << matrix[i * shape[1] + j] << " ";
			}
			std::cout << "\n";
		}
	}

	std::unique_ptr<Matrix> reshape(int rows, int cols) {
		if (rows * cols != *size) throw std::invalid_argument("New matrix size does not match original matrix size!");

		std::unique_ptr<int[]> new_shape = std::make_unique<int[]>(2);
		new_shape[0] = rows;
		new_shape[1] = cols;

		std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(matrix, new_shape);

		return ret_matrix;
	}

	// This could be done in parallel on the GPU
	std::unique_ptr<Matrix> transpose() {
		std::unique_ptr<int[]> new_shape = std::make_unique<int[]>(2);
		new_shape[0] = shape[1];
		new_shape[1] = shape[0];

		int bytes = *size * sizeof(float);

		float* dCopy;
		hipMalloc(&dCopy, bytes);
		hipMemcpy(dCopy, matrix.get(), bytes, hipMemcpyHostToDevice);

		int blockSizeCols = (shape[0] + NUM_THREADS - 1) / NUM_THREADS;
		int blockSizeRows = (shape[1] + NUM_THREADS - 1) / NUM_THREADS;
		dim3 THREADS(NUM_THREADS_BLOCK, NUM_THREADS_BLOCK);
		dim3 BLOCKS(blockSizeCols, blockSizeRows);

		transposeD <<< BLOCKS, THREADS >>> (shape[0], shape[1], dCopy);

		std::unique_ptr<float[]> new_matrix = std::make_unique<float[]>(*size);
		hipMemcpy(new_matrix.get(), dCopy, bytes, hipMemcpyDeviceToHost);

		std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(new_matrix, new_shape);

		hipFree(dCopy);

		return ret_matrix;
	}

	std::unique_ptr<Matrix> clone() {
		std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(matrix, shape);
		return ret_matrix;
	}

	// I cant run this on my GPU yet for some reason so it has to be done on the CPU I will get it working though
	template <typename Lambda>
	std::unique_ptr<Matrix> apply(Lambda function) {
		int bytes = *size * sizeof(float);

		float* dCopy;
		hipMalloc(&dCopy, bytes);
		hipMemcpy(dCopy, matrix.get(), bytes, hipMemcpyHostToDevice);

		applyD <<< 1, NUM_THREADS >>> (*size, dCopy, function);

		std::unique_ptr<float[]> new_matrix = std::make_unique<float[]>(*size);
		hipMemcpy(new_matrix.get(), dCopy, bytes, hipMemcpyDeviceToHost);

		std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(new_matrix, shape);

		hipFree(dCopy);

		return ret_matrix;
	}

	std::unique_ptr<float[]> returnMatrix() {
		std::unique_ptr<float[]> ret_matrix = std::make_unique<float[]>(*size);
		memcpy(ret_matrix.get(), matrix.get(), *size * sizeof(float));

		return ret_matrix;
	}

	std::unique_ptr<int[]> returnShape() {
		std::unique_ptr<int[]> ret_shape = std::make_unique<int[]>(2);
		memcpy(ret_shape.get(), shape.get(), 2 * sizeof(unsigned int));
		return ret_shape;
	}

	int returnSize() {
		return *size;
	}
};

__global__
void addD(int size, float* vector1, float *vector2, float *retVector) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < size) retVector[idx] = vector1[idx] + vector2[idx];
}

// For subtraction just multiply the array by 1 the add into a negative which can be done through the apply func
std::unique_ptr<Matrix> add(std::unique_ptr<Matrix> &matrix1, std::unique_ptr<Matrix> &matrix2) {
	std::unique_ptr<int[]> mat1shape = matrix1->returnShape();
	std::unique_ptr<int[]> mat2shape = matrix2->returnShape();
	if ((mat1shape[0] != mat2shape[0]) || (mat1shape[1] != mat2shape[1])) throw std::invalid_argument("Matrices are not of the same shape!");

	int size = matrix1->returnSize();
	int bytes = size * sizeof(float);

	std::unique_ptr<float[]> mat1 = matrix1->returnMatrix();
	std::unique_ptr<float[]> mat2 = matrix2->returnMatrix();

	float* mat1d;
	float* mat2d;
	float* mat3d;
	hipMalloc(&mat1d, bytes);
	hipMalloc(&mat2d, bytes);
	hipMalloc(&mat3d, bytes);
	hipMemcpy(mat1d, mat1.get(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(mat2d, mat2.get(), bytes, hipMemcpyHostToDevice);

	addD <<< 1, NUM_THREADS >>> (size, mat1d, mat2d, mat3d);

	std::unique_ptr<float[]> mat3 = std::make_unique<float[]>(bytes);
	hipMemcpy(mat3.get(), mat3d, bytes, hipMemcpyDeviceToHost);

	std::unique_ptr<int[]> shape = matrix1->returnShape();
	std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(mat3, shape);

	hipFree(mat1d);
	hipFree(mat2d);
	hipFree(mat3d);

	return ret_matrix;
}

__global__
// What are the specifications required for matrix multiplication...?
void multiplyD(int rows, int same, int cols, float* vector1, float* vector2, float* retVector) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if ((row < rows) && (col < cols)) {
		float sum = 0;
		for (int i = 0; i < same; i++) {
			sum += vector1[row * same + i] * vector2[i * cols + col];
		}
		retVector[row * cols + col] = sum;
	}
}

std::unique_ptr<Matrix> multiply(std::unique_ptr<Matrix> &matrix1, std::unique_ptr<Matrix> &matrix2) {
	std::unique_ptr<int[]> mat1shape = matrix1->returnShape();
	std::unique_ptr<int[]> mat2shape = matrix2->returnShape();
	if (mat1shape[1] != mat2shape[0]) throw std::invalid_argument("Matrix1's cols must equal Matrix2's rows!");

	std::unique_ptr<int[]> new_shape = std::make_unique<int[]>(2);
	new_shape[0] = mat1shape[0];
	new_shape[1] = mat2shape[1];
	int same = mat1shape[1];

	int mat1bytes = matrix1->returnSize() * sizeof(float);
	int mat2bytes = matrix2->returnSize() * sizeof(float);
	int mat3bytes = new_shape[0] * new_shape[1] * sizeof(float);

	float* mat1d;
	float* mat2d;
	float* mat3d;
	hipMalloc(&mat1d, mat1bytes);
	hipMalloc(&mat2d, mat2bytes);
	hipMalloc(&mat3d, mat3bytes);

	std::unique_ptr<float[]> mat1 = matrix1->returnMatrix();
	std::unique_ptr<float[]> mat2 = matrix2->returnMatrix();
	hipMemcpy(mat1d, mat1.get(), mat1bytes, hipMemcpyHostToDevice);
	hipMemcpy(mat2d, mat2.get(), mat2bytes, hipMemcpyHostToDevice);

	int blockSizeCols = (new_shape[0] + NUM_THREADS - 1) / NUM_THREADS;
	int blockSizeRows = (new_shape[1] + NUM_THREADS - 1) / NUM_THREADS;
	dim3 THREADS(NUM_THREADS_BLOCK, NUM_THREADS_BLOCK);
	dim3 BLOCKS(blockSizeCols, blockSizeRows);

	multiplyD <<< BLOCKS, THREADS >>> (new_shape[0], same, new_shape[1], mat1d, mat2d, mat3d);

	std::unique_ptr<float[]> mat3 = std::make_unique<float[]>(new_shape[0] * new_shape[1]);
	hipMemcpy(mat3.get(), mat3d, mat3bytes, hipMemcpyDeviceToHost);

	std::unique_ptr<Matrix> ret_matrix = std::make_unique<Matrix>(mat3, new_shape);

	hipFree(mat1d);
	hipFree(mat2d);
	hipFree(mat3d);

	return ret_matrix;
}

// Make some functions which can initialize matrices for us now

int main() {
	std::unique_ptr<int[]> shape1 = std::make_unique<int[]>(2);
	shape1[0] = 5;
	shape1[1] = 10;
	std::unique_ptr<float[]> vals1 = std::make_unique<float[]>(10);
	for (int i = 0; i < 10; i++) {
		vals1[i] = 2.0f;
	}
	std::unique_ptr<Matrix> matrix1 = std::make_unique<Matrix>(vals1, shape1);

	std::unique_ptr<int[]> shape2 = std::make_unique<int[]>(2);
	shape2[0] = 10;
	shape2[1] = 4;
	std::unique_ptr<float[]> vals2 = std::make_unique<float[]>(10);
	for (int i = 0; i < 10; i++) {
		vals2[i] = 3.0f;
	}
	std::unique_ptr<Matrix> matrix2 = std::make_unique<Matrix>(vals2, shape2);

	// This error means there must be something wrong with our memory allocation for the block count or possibly the row/col allocation
	std::unique_ptr<Matrix> transposed = matrix2->transpose();
	transposed->print();

//	std::unique_ptr<Matrix> multiplied = multiply(matrix1, matrix2);
//	multiplied->print();
}
